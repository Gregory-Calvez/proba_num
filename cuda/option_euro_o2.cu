#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <time.h>

const int num_blocks = 1024;
const int num_threads = 256;
const int num_iterations = 10;
const int num_steps = 100;

__global__ void setup_states(hiprandState* states){
    int id = threadIdx.x + num_threads * blockIdx.x;
    // Initialisation states
    hiprand_init(0, id, 0, &states[id]);
}

__global__ void put_euro_heston_cir_o2(hiprandState* states, float* results_sum, float* results_sum_squared, float x_0, float cir_0, float r, float a, float k, float sigma, float rho, float t, float strike)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Saving the state in the GPU memory to be more efficient
    hiprandState localState = states[id];
    // Shared memory for the Monte Carlo
    __shared__ float partial_sums[num_threads];
    __shared__ float partial_sums_squared[num_threads];

    // Some values that we will not have to compute twice / maybe we should give this in parameters.
    float x_1;
    float x_2;
    float x_3;
    float x_4;
    float dx_1;
    float dt = t / num_steps;
    float u_1; // u will be a uniform variable
    float u_2; //
    float y;
    float2 n; // n will be a pair of normal variable
    float u_tilde_1;
    float u_tilde_2;
    float pi;
    float value_option;

    float aux_hz_1 = (1 - rho*rho) * dt;
    float aux_hw_1 = (r - rho * a / sigma) * dt;
    float aux_hw_2 = rho / sigma;
    float aux_hw_3 = (rho*k/sigma - 0.5) * dt;
    float srqt3 = sqrtf(3);
    float aux_phi_1;

    float aux_sigma2sur4moinsa = sigma*sigma / 4 - a;
    float psi_k;
    if (k == 0){
        psi_k = dt / 2;
    }
    else {
        psi_k = ( 1 - expf(-k * dt / 2) ) / k;
    };
    float k_2;
    float expo = expf(k * dt / 2);
    float expo_2 = expf(-k*dt);
    float aux_k_2_1 = sqrtf(expo * aux_sigma2sur4moinsa * psi_k) + sigma / 2 * sqrt(3*t);
    if (aux_sigma2sur4moinsa > 0){
        k_2 = expo * (aux_sigma2sur4moinsa * psi_k + aux_k_2_1 * aux_k_2_1);
    }
    else {
        k_2 = 0;
    };

    float psi_k_2;
    if (k == 0){
        psi_k_2 = dt;
    }
    else {
        psi_k_2 = ( 1 - expf(-k * dt) ) / k;
    };
    float aux_u_tilde_1 = a*psi_k_2;

    // Main loop for the Monte Carlo
    for(unsigned int i = 0; i < num_iterations; ++i){
        // Initialization of the shared memory at the begining of the MC
        if(i == 0){
            partial_sums[threadIdx.x] = 0;
            partial_sums_squared[threadIdx.x] = 0;
        };
        x_1 = cir_0;  // Vol process
        x_2 = 0;  // Integration of the vol process
        x_3 = x_0;  // Stock process
        x_4 = 0;  // Integration of the stock process

        // CIR_O2 and Heston
        for (unsigned int k = 0; k < num_steps; ++k){
            // printf("Value of the Heston %f  and the CIR %f at the step %d. \n", x_3, x_1, k);
            u_1 = hiprand_uniform(&localState);
            n = hiprand_normal2(&localState); // It is not optimal, we simulate two uniform for 1 normal
            if (u_1 < 0.5) {
                // HZ
                x_3 = x_3 * expf(sqrtf(x_1 * aux_hz_1)*n.x);
                // HW
                dx_1 = - x_1;
/////////////// CIR_O2
                if(x_1 >= k_2){
                    u_2 = hiprand_uniform(&localState);
                    if(u_2 < 1./6.){
                        y = - srqt3;
                    }
                    else {
                        if (u_2 < 5./6.){
                            y = 0;
                        }
                        else {
                            y = srqt3;
                        };
                    };
                    aux_phi_1 = sqrtf( - aux_sigma2sur4moinsa * psi_k + x_1 /expo ) + sigma / 2 * sqrtf(dt) * y;
                    x_1 = 1 / expo * aux_phi_1 * aux_phi_1 - aux_sigma2sur4moinsa * psi_k;
                }
                else {
                    u_tilde_1 = x_1 * expo_2 + aux_u_tilde_1;
                    u_tilde_2 = u_tilde_1 * u_tilde_1 + sigma * sigma * psi_k_2 * (a * psi_k_2 / 2.0f + x_1 * expo_2);
                    pi = 0.5f * (1 - sqrtf(1 - u_tilde_1 * u_tilde_1 / u_tilde_2) );
                    u_2 = hiprand_uniform(&localState);
                    if (u_2 < pi){
                        x_1 = u_tilde_1 / 2.0f / pi;
                    }
                    else {
                        x_1 = u_tilde_1 / 2.0f / (1.0f - pi);
                    };
                };
/////////////// Fin CIR_O2

                dx_1 += x_1;
                x_2 += (x_1 - 0.5*dx_1) * dt;
                x_4 += 0.5*x_3*dt;
                x_3 = x_3 * expf( aux_hw_1 + aux_hw_2 * dx_1 + aux_hw_3 * (x_1 - 0.5 * dx_1) );
                x_4 += 0.5*x_3*dt;
            }
            else {
/////////////// HW
                dx_1 = - x_1;
/////////////// CIR_O2
                if(x_1 >= k_2){
                    u_2 = hiprand_uniform(&localState);
                    if(u_2 < 1./6.){
                        y = - srqt3;
                    }
                    else {
                        if (u_2 < 5./6.){
                            y = 0;
                        }
                        else {
                            y = srqt3;
                        };
                    };
                    aux_phi_1 = sqrtf( - aux_sigma2sur4moinsa * psi_k + x_1 /expo ) + sigma / 2 * sqrtf(dt) * y;
                    x_1 = 1 / expo * aux_phi_1 * aux_phi_1 - aux_sigma2sur4moinsa * psi_k;
                }
                else {
                    u_tilde_1 = x_1 * expo_2 + aux_u_tilde_1;
                    u_tilde_2 = u_tilde_1 * u_tilde_1 + sigma * sigma * psi_k_2 * (a * psi_k_2 / 2.0f + x_1 * expo_2);
                    pi = 0.5f * (1 - sqrtf(1 - u_tilde_1 * u_tilde_1 / u_tilde_2) );
                    u_2 = hiprand_uniform(&localState);
                    if (u_2 < pi){
                        x_1 = u_tilde_1 / 2.0f / pi;
                    }
                    else {
                        x_1 = u_tilde_1 / 2.0f / (1.0f - pi);
                    };
                };
/////////////// Fin CIR_O2
                dx_1 += x_1;
                x_2 += (x_1 - 0.5*dx_1) * dt;
                x_4 += 0.5*x_3*dt;
                x_3 = x_3 * expf( aux_hw_1 + aux_hw_2 * dx_1 + aux_hw_3 * (x_1 - 0.5 * dx_1) );
                x_4 += 0.5*x_3*dt;
/////////////// HZ
                x_3 = x_3 * expf(sqrtf(x_1 * aux_hz_1)*n.x);
            }
        };
        value_option = fmaxf(0, strike - x_3);
        partial_sums[threadIdx.x] += value_option;
        partial_sums_squared[threadIdx.x] += value_option * value_option;
        // printf("%f\n", value_option);
    };

    // Synchronize the threads
    __syncthreads();

    // Sum per block
    if(threadIdx.x == 0){
        float sum = 0;
        float sum_squared = 0.0f;
        for (int i = 0; i < blockDim.x; ++i){
            sum += partial_sums[i];
            sum_squared += partial_sums_squared[i];
        };
        results_sum[blockIdx.x] += sum;
        results_sum_squared[blockIdx.x] += sum_squared;
    };

    // Saving the states in the global memory
    states[id] = localState;
};


long timediff(clock_t t1, clock_t t2) {
    long elapsed;
    elapsed = ((double)t2 - t1) / CLOCKS_PER_SEC * 1000;
    return elapsed;
};


int main(void){

    clock_t t1;
    clock_t t2;
    long elapsed;

    float k = 0.5f;
    float a = 0.02f;
    float sigma = 0.4f;
    float x_0 = 100.0f;
    float cir_0 = 0.04;
    float rho = - 0.5f;
    float r = 0.02f;
    float strike = 100.0f;
    float expiry = 1.0f;


    t1 = clock();

    float *h_results_sum, *d_results_sum;
    h_results_sum = (float*)malloc(num_blocks * sizeof(float));
    hipMalloc(&d_results_sum, num_blocks * sizeof(float));
    for(int i = 0; i < num_blocks; ++i){
        h_results_sum[i] = 0;
    };
    float *h_results_sum_squared, *d_results_sum_squared;
    h_results_sum_squared = (float*)malloc(num_blocks * sizeof(float));
    hipMalloc(&d_results_sum_squared, num_blocks * sizeof(float));
    for(int i = 0; i < num_blocks; ++i){
        h_results_sum_squared[i] = 0;
    };
    hipMemcpy(d_results_sum, h_results_sum, num_blocks * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_results_sum_squared, h_results_sum_squared, num_blocks * sizeof(float), hipMemcpyHostToDevice);

    hiprandState *d_states;
    hipMalloc(&d_states, num_threads * num_blocks * sizeof(hiprandState));

    setup_states <<< num_blocks, num_threads >>> (d_states);
    hipDeviceSynchronize();
    put_euro_heston_cir_o2 <<< num_blocks, num_threads >>> (d_states, d_results_sum, d_results_sum_squared, x_0, cir_0, r, a, k, sigma, rho, expiry, strike);
    hipDeviceSynchronize();

    hipMemcpy(h_results_sum, d_results_sum, num_blocks * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_results_sum_squared, d_results_sum_squared, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Global sum
    float global_results_sum = 0;
    for(int i = 0; i < num_blocks; i++){
        global_results_sum += h_results_sum[i];
    };
    global_results_sum = global_results_sum * expf(-r * expiry);
    float global_results_sum_squared = 0;
    for(int i = 0; i < num_blocks; i++){
        global_results_sum_squared += h_results_sum_squared[i];
    };
    global_results_sum_squared = global_results_sum_squared * expf(- 2 * r * expiry);
    unsigned int num_simu = (num_blocks * num_threads * num_iterations);
    float empirical_expectency = global_results_sum / num_simu;
    float empirical_squared = global_results_sum_squared / num_simu;
    float empirical_variance = empirical_squared - empirical_expectency * empirical_expectency;
    float confidence_interval_low = empirical_expectency - 1.96 * sqrtf(empirical_variance / num_simu);
    float confidence_interval_high = empirical_expectency + 1.96 * sqrtf(empirical_variance / num_simu);

    t2 = clock();

    printf("We have computed a MC Call price of : %f\n", empirical_expectency);
    printf("Empirical variance : %f\n", empirical_variance);
    printf("Number of simulations: %d\n", num_simu);
    printf("Confidence interval : (%f , %f)\n", confidence_interval_low, confidence_interval_high);
    elapsed = timediff(t1, t2);
    printf("Time elapsed: %ld ms\n", elapsed);

    return 0;
}
