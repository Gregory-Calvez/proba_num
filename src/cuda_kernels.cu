#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <time.h>

const int num_blocks = 1024;
const int num_threads = 256;
const int num_iterations = 10;


__global__ void setup_states(hiprandState* states){
    int id = threadIdx.x + num_threads * blockIdx.x;
    // Initialisation states
    hiprand_init(0, id, 0, &states[id]);
}

__global__ void put_euro_heston_cir_o2(hiprandState* states, float* results_sum, float* results_sum_squared, float x_0, float cir_0, float r, float a, float k, float sigma, float rho, float t, float strike, unsigned int num_steps, char type)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Saving the state in the GPU memory to be more efficient
    hiprandState localState = states[id];
    // Shared memory for the Monte Carlo
    __shared__ float partial_sums[num_threads];
    __shared__ float partial_sums_squared[num_threads];

    // Some values that we will not have to compute twice / maybe we should give this in parameters.
    float x_1;
    float x_2;
    float x_3;
    float x_4;
    float dx_1;
    float dt = t / num_steps;
    float u_1; // u will be a uniform variable
    float u_2; //
    float y;
    float2 n; // n will be a pair of normal variable
    float u_tilde_1;
    float u_tilde_2;
    float pi;
    float value_option;

    float aux_hz_1 = (1 - rho*rho) * dt;
    float aux_hw_1 = (r - rho * a / sigma) * dt;
    float aux_hw_2 = rho / sigma;
    float aux_hw_3 = (rho*k/sigma - 0.5) * dt;
    float srqt3 = sqrtf(3);
    float aux_phi_1;

    float aux_sigma2sur4moinsa = sigma*sigma / 4 - a;
    float psi_k;
    if (k == 0){
        psi_k = dt / 2;
    }
    else {
        psi_k = ( 1 - expf(-k * dt / 2) ) / k;
    };
    float k_2;
    float expo = expf(k * dt / 2);
    float expo_2 = expf(-k*dt);
    float aux_k_2_1 = sqrtf(expo * aux_sigma2sur4moinsa * psi_k) + sigma / 2 * sqrt(3*t);
    if (aux_sigma2sur4moinsa > 0){
        k_2 = expo * (aux_sigma2sur4moinsa * psi_k + aux_k_2_1 * aux_k_2_1);
    }
    else {
        k_2 = 0;
    };

    float psi_k_2;
    if (k == 0){
        psi_k_2 = dt;
    }
    else {
        psi_k_2 = ( 1 - expf(-k * dt) ) / k;
    };
    float aux_u_tilde_1 = a*psi_k_2;

    // Main loop for the Monte Carlo
    for(unsigned int i = 0; i < num_iterations; ++i){
        // Initialization of the shared memory at the begining of the MC
        if(i == 0){
            partial_sums[threadIdx.x] = 0;
            partial_sums_squared[threadIdx.x] = 0;
        };
        x_1 = cir_0;  // Vol process
        x_2 = 0;  // Integration of the vol process
        x_3 = x_0;  // Stock process
        x_4 = 0;  // Integration of the stock process

        // CIR_O2 and Heston
        for (unsigned int k = 0; k < num_steps; ++k){
            // printf("Value of the Heston %f  and the CIR %f at the step %d. \n", x_3, x_1, k);
            u_1 = hiprand_uniform(&localState);
            n = hiprand_normal2(&localState); // It is not optimal, we simulate two uniform for 1 normal
            if (u_1 < 0.5) {
                // HZ
                x_3 = x_3 * expf(sqrtf(x_1 * aux_hz_1)*n.x);
                // HW
                dx_1 = - x_1;
/////////////// CIR_O2
                if(x_1 >= k_2){
                    u_2 = hiprand_uniform(&localState);
                    if(u_2 < 1./6.){
                        y = - srqt3;
                    }
                    else {
                        if (u_2 < 5./6.){
                            y = 0;
                        }
                        else {
                            y = srqt3;
                        };
                    };
                    aux_phi_1 = sqrtf( - aux_sigma2sur4moinsa * psi_k + x_1 /expo ) + sigma / 2 * sqrtf(dt) * y;
                    x_1 = 1 / expo * aux_phi_1 * aux_phi_1 - aux_sigma2sur4moinsa * psi_k;
                }
                else {
                    u_tilde_1 = x_1 * expo_2 + aux_u_tilde_1;
                    u_tilde_2 = u_tilde_1 * u_tilde_1 + sigma * sigma * psi_k_2 * (a * psi_k_2 / 2.0f + x_1 * expo_2);
                    pi = 0.5f * (1 - sqrtf(1 - u_tilde_1 * u_tilde_1 / u_tilde_2) );
                    u_2 = hiprand_uniform(&localState);
                    if (u_2 < pi){
                        x_1 = u_tilde_1 / 2.0f / pi;
                    }
                    else {
                        x_1 = u_tilde_1 / 2.0f / (1.0f - pi);
                    };
                };
/////////////// Fin CIR_O2

                dx_1 += x_1;
                x_2 += (x_1 - 0.5*dx_1) * dt;
                x_4 += 0.5*x_3*dt;
                x_3 = x_3 * expf( aux_hw_1 + aux_hw_2 * dx_1 + aux_hw_3 * (x_1 - 0.5 * dx_1) );
                x_4 += 0.5*x_3*dt;
            }
            else {
/////////////// HW
                dx_1 = - x_1;
/////////////// CIR_O2
                if(x_1 >= k_2){
                    u_2 = hiprand_uniform(&localState);
                    if(u_2 < 1./6.){
                        y = - srqt3;
                    }
                    else {
                        if (u_2 < 5./6.){
                            y = 0;
                        }
                        else {
                            y = srqt3;
                        };
                    };
                    aux_phi_1 = sqrtf( - aux_sigma2sur4moinsa * psi_k + x_1 /expo ) + sigma / 2 * sqrtf(dt) * y;
                    x_1 = 1 / expo * aux_phi_1 * aux_phi_1 - aux_sigma2sur4moinsa * psi_k;
                }
                else {
                    u_tilde_1 = x_1 * expo_2 + aux_u_tilde_1;
                    u_tilde_2 = u_tilde_1 * u_tilde_1 + sigma * sigma * psi_k_2 * (a * psi_k_2 / 2.0f + x_1 * expo_2);
                    pi = 0.5f * (1 - sqrtf(1 - u_tilde_1 * u_tilde_1 / u_tilde_2) );
                    u_2 = hiprand_uniform(&localState);
                    if (u_2 < pi){
                        x_1 = u_tilde_1 / 2.0f / pi;
                    }
                    else {
                        x_1 = u_tilde_1 / 2.0f / (1.0f - pi);
                    };
                };
/////////////// Fin CIR_O2
                dx_1 += x_1;
                x_2 += (x_1 - 0.5*dx_1) * dt;
                x_4 += 0.5*x_3*dt;
                x_3 = x_3 * expf( aux_hw_1 + aux_hw_2 * dx_1 + aux_hw_3 * (x_1 - 0.5 * dx_1) );
                x_4 += 0.5*x_3*dt;
/////////////// HZ
                x_3 = x_3 * expf(sqrtf(x_1 * aux_hz_1)*n.x);
            }
        };
        if(type == 'e'){
            value_option = fmaxf(0, strike - x_3);
        }
        else {
            value_option = fmaxf(0, strike - x_4);
        }
        partial_sums[threadIdx.x] += value_option;
        partial_sums_squared[threadIdx.x] += value_option * value_option;
        // printf("%f\n", value_option);
    };

    // Synchronize the threads
    __syncthreads();

    // Sum per block
    if(threadIdx.x == 0){
        float sum = 0;
        float sum_squared = 0.0f;
        for (int i = 0; i < blockDim.x; ++i){
            sum += partial_sums[i];
            sum_squared += partial_sums_squared[i];
        };
        results_sum[blockIdx.x] += sum;
        results_sum_squared[blockIdx.x] += sum_squared;
    };

    // Saving the states in the global memory
    states[id] = localState;
};


long timediff(clock_t t1, clock_t t2) {
    long elapsed;
    elapsed = ((double)t2 - t1) / CLOCKS_PER_SEC * 1000;
    return elapsed;
};


void wrapper_kernel_o2(float* output, float x_0, float cir_0, float r, float a, float k, float sigma, float rho, float expiry, float strike, unsigned int num_steps, char type){
    clock_t t1;
    clock_t t2;
    long elapsed;

    t1 = clock();
    float *h_results_sum, *d_results_sum;
    h_results_sum = (float*)malloc(num_blocks * sizeof(float));
    hipMalloc(&d_results_sum, num_blocks * sizeof(float));
    for(int i = 0; i < num_blocks; ++i){
        h_results_sum[i] = 0;
    };
    float *h_results_sum_squared, *d_results_sum_squared;
    h_results_sum_squared = (float*)malloc(num_blocks * sizeof(float));
    hipMalloc(&d_results_sum_squared, num_blocks * sizeof(float));
    for(int i = 0; i < num_blocks; ++i){
        h_results_sum_squared[i] = 0;
    };
    hipMemcpy(d_results_sum, h_results_sum, num_blocks * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_results_sum_squared, h_results_sum_squared, num_blocks * sizeof(float), hipMemcpyHostToDevice);

    hiprandState *d_states;
    hipMalloc(&d_states, num_threads * num_blocks * sizeof(hiprandState));

    setup_states <<< num_blocks, num_threads >>> (d_states);
    hipDeviceSynchronize();
    put_euro_heston_cir_o2 <<< num_blocks, num_threads >>> (d_states, d_results_sum, d_results_sum_squared, x_0, cir_0, r, a, k, sigma, rho, expiry, strike, num_steps, type);
    hipDeviceSynchronize();

    hipMemcpy(h_results_sum, d_results_sum, num_blocks * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_results_sum_squared, d_results_sum_squared, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Global sum
    float global_results_sum = 0;
    for(int i = 0; i < num_blocks; i++){
        global_results_sum += h_results_sum[i];
    };
    global_results_sum = global_results_sum * expf(-r * expiry);
    float global_results_sum_squared = 0;
    for(int i = 0; i < num_blocks; i++){
        global_results_sum_squared += h_results_sum_squared[i];
    };
    global_results_sum_squared = global_results_sum_squared * expf(- 2 * r * expiry);
    unsigned int num_simu = (num_blocks * num_threads * num_iterations);
    float empirical_expectency = global_results_sum / num_simu;
    float empirical_squared = global_results_sum_squared / num_simu;
    float empirical_variance = empirical_squared - empirical_expectency * empirical_expectency;
    float confidence_interval_low = empirical_expectency - 1.96 * sqrtf(empirical_variance / num_simu);
    float confidence_interval_high = empirical_expectency + 1.96 * sqrtf(empirical_variance / num_simu);

    t2 = clock();
    elapsed = timediff(t1, t2);

    output[0] = empirical_expectency;
    output[1] = confidence_interval_low;
    output[2] = elapsed;

    printf("We have computed a MC Call price of : %f\n", empirical_expectency);
    printf("Empirical variance : %f\n", empirical_variance);
    printf("Number of simulations: %d\n", num_simu);
    printf("Confidence interval : (%f , %f)\n", confidence_interval_low, confidence_interval_high);
    printf("Time elapsed: %ld ms\n", elapsed);

    return;
};


__global__ void put_heston_cir_o3(hiprandState* states, float* results_sum, float* results_sum_squared, float x_0, float cir_0, float r, float a, float k, float sigma, float rho, float t, float strike, unsigned int num_steps, char type)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Saving the state in the GPU memory to be more efficient
    hiprandState localState = states[id];
    // Shared memory for the Monte Carlo
    __shared__ float partial_sums[num_threads];
    __shared__ float partial_sums_squared[num_threads];

    // Some values that we will not have to compute twice / maybe we should give this in parameters.
    float x_1;
    float x_2;
    float x_3;
    float x_4;
    float dx_1;
    float dt = t / num_steps;
    float u_1; // u will be a uniform variable
    float u_2; // idem
    float u_3; // idem
    float2 n; // n will be a pair of normal variable
    float y;
    float epsilon;
    float u_tilde_1;
    float u_tilde_2;
    float u_tilde_3;
    float s;
    float p;
    float delta;
    float pi;
    float value_option;

    float sqrt3 = sqrtf(3.);
    float sqrt2 = sqrtf(2.);
    float s_3_m_s6 = sqrtf( 3 - sqrtf(6));
    float s_3_p_s6 = sqrtf( 3 + sqrtf(6));
    float aux_proba_y = (sqrtf(6.) - 2.)/ (4.0f*sqrtf(6.));

    float sigma_2 = sigma * sigma;
    float four_a_over_3 = 4*a / 3;
    float four_a = 4*a;
    float sigma_2_over_4_minus_a = sigma_2 / 4 - a;
    float sigma_2_over_4_minus_a_abs;
    if (sigma_2_over_4_minus_a > 0){
        sigma_2_over_4_minus_a_abs = sigma_2_over_4_minus_a;
    } else {
        sigma_2_over_4_minus_a_abs = -sigma_2_over_4_minus_a;
    };
    float aux_hz_1 = (1. - rho*rho) * dt;
    float aux_hw_1 = (r - rho * a / sigma) * dt;
    float aux_hw_2 = rho / sigma;
    float aux_hw_3 = (rho*k/sigma - 0.5) * dt;

    float psi_k;
    if (k == 0){
        psi_k = dt ;
    }
    else {
        psi_k = ( 1 - expf(-k * dt) ) / k;
    };
    float psi_minus_k;
    if (k == 0){
        psi_minus_k = dt;
    }
    else {
        psi_minus_k = ( 1 - expf(+k * dt) ) / (-k);
    };

    float k_3 = 0;
    float aux_k_3_1 = sqrtf(sigma_2 / 4 - a + sigma / sqrtf(2) * sqrtf(-sigma_2_over_4_minus_a)) + sigma / 2 * s_3_p_s6;
    float aux_k_3_2 = sigma / sqrt2 * sqrtf(-sigma_2_over_4_minus_a);
    float aux_k_3_3 = sqrtf(sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a)) + sigma / 2 * s_3_p_s6;
    float aux_k_3_4 = sigma_2 / 4 - a + aux_k_3_3 * aux_k_3_3;
    if (sigma_2 <= four_a_over_3) {
        k_3 = aux_k_3_2;
    } else if (sigma_2 <= four_a) {
        k_3 = aux_k_3_1 * aux_k_3_1;
    } else {
        k_3 = aux_k_3_4;
    };
    k_3 *= psi_minus_k;
    float expo = expf(k * dt / 2);
    float expo_2 = expf(-k*dt);
    float aux_u_tilde_1 = a*psi_k;
    float aux_u_tilde_3_1 = psi_k * (a + sigma_2 / 2) ;
    float aux_u_tilde_3_2 = 2*expf(-2*k*dt);

    // Main loop for the Monte Carlo
    for(unsigned int i = 0; i < num_iterations; ++i){
        // Initialization of the shared memory at the begining of the MC
        if(i == 0){
            partial_sums[threadIdx.x] = 0;
            partial_sums_squared[threadIdx.x] = 0;
        };
        x_1 = cir_0;  // Vol process
        x_2 = 0;  // Integration of the vol process
        x_3 = x_0;  // Stock process
        x_4 = 0;  // Integration of the stock process

        // CIR_O3 and Heston
        for (unsigned int k = 0; k < num_steps; ++k){
            // printf("Value of the Heston %f  and the CIR %f at the step %d. \n", x_3, x_1, k);
            u_1 = hiprand_uniform(&localState);
            n = hiprand_normal2(&localState); // It is not optimal, we simulate two uniform for 1 normal
            if (u_1 < 0.5) {
                // HZ
                x_3 = x_3 * expf(sqrtf(x_1 * aux_hz_1)*n.x);
                // HW
                dx_1 = - x_1;
/////////////// CIR_O3
                if(x_1 >= k_3){
                    u_1 = hiprand_uniform(&localState);
                    u_2 = hiprand_uniform(&localState);
                    u_3 = hiprand_uniform(&localState);
                    // Computing y
                    if (u_1 < aux_proba_y) {
                        y = - s_3_p_s6;
                    } else if (u_1 < 2. * aux_proba_y){
                        y = + s_3_p_s6;
                    } else if (u_1 < 0.5 + aux_proba_y){
                        y = - s_3_m_s6;
                    } else {
                        y = + s_3_m_s6;
                    };
                    // Computing epsilon
                    if (u_2 < 1./2.){
                        epsilon = -1.;
                    } else {
                        epsilon = +1.;
                    };
                    // zeta
                    if (u_3 < 1./3.) {
                        if (sigma_2_over_4_minus_a <= 0) {
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                        } else{
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                        };
                    } else if (u_3 < 2./3.){
                        if (sigma_2_over_4_minus_a <= 0) {
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2.0f) * fmaxf(0.0f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                        } else{
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 = fmaxf(0.0f,  sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2.0f) * fmaxf(0.0f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                        };
                    } else {
                        if (sigma_2_over_4_minus_a <= 0) {
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 = fmaxf(0.0f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2.f) * fmaxf(0.0f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                        } else{
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 = fmaxf(0.0f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2.0f) * fmaxf(0.0f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                        };
                    };
                    x_1 *= expo_2;
                } else {
                    u_1 = hiprand_uniform(&localState);
                    u_tilde_1 = x_1 * expo_2 + aux_u_tilde_1;
                    u_tilde_2 = u_tilde_1 * u_tilde_1 + sigma * sigma * psi_k * (a * psi_k  / 2.0f + x_1 * expo_2);
                    u_tilde_3 = u_tilde_1 * u_tilde_2 + sigma_2 * psi_k * (x_1*x_1 * aux_u_tilde_3_2 + aux_u_tilde_3_1* (3*x_1 * expo_2 + a * psi_k));
                    s = (u_tilde_3 - u_tilde_1 * u_tilde_2) / (u_tilde_2 - u_tilde_1 * u_tilde_1);
                    p = (u_tilde_1 * u_tilde_3 - u_tilde_2*u_tilde_2) / (u_tilde_2 - u_tilde_1*u_tilde_1);
                    delta = sqrtf(s*s - 4.*p);
                    pi = (u_tilde_1 - (s-delta) / 2.f) / delta;
                    if (u_1 < pi) {
                        x_1 = (s + delta) / 2.f;
                    } else {
                        x_1 = (s - delta) / 2.f;
                    };
                };
/////////////// Fin CIR_O3
                dx_1 += x_1;
                x_2 += (x_1 - 0.5*dx_1) * dt;
                x_4 += 0.5*x_3*dt;
                x_3 = x_3 * expf( aux_hw_1 + aux_hw_2 * dx_1 + aux_hw_3 * (x_1 - 0.5 * dx_1) );
                x_4 += 0.5*x_3*dt;
            }
            else {
                // HW
                dx_1 = - x_1;
/////////////// CIR_O3
                if(x_1 >= k_3){
                    u_1 = hiprand_uniform(&localState);
                    u_2 = hiprand_uniform(&localState);
                    u_3 = hiprand_uniform(&localState);
                    // Computing y
                    if (u_1 < aux_proba_y) {
                        y = - s_3_p_s6;
                    } else if (u_1 < 2. * aux_proba_y){
                        y = + s_3_p_s6;
                    } else if (u_1 < 0.5 + aux_proba_y){
                        y = - s_3_m_s6;
                    } else {
                        y = + s_3_m_s6;
                    };
                    // Computing epsilon
                    if (u_2 < 1./2.){
                        epsilon = -1.;
                    } else {
                        epsilon = +1.;
                    };
                    // zeta
                    if (u_3 < 1./3.) {
                        if (sigma_2_over_4_minus_a <= 0) {
                            x_1 = fmaxf(0.f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2.f) * fmaxf(0.f, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                        } else{
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                        };
                    } else if (u_3 < 2./3.){
                        if (sigma_2_over_4_minus_a <= 0) {
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                        } else{
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                        };
                    } else {
                        if (sigma_2_over_4_minus_a <= 0) {
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                        } else{
                            x_1 += sigma / sqrt2 * sqrtf(sigma_2_over_4_minus_a_abs) * epsilon * psi_minus_k;  // Xt
                            x_1 += -sigma_2_over_4_minus_a * psi_minus_k;  // X0
                            x_1 = fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2) * fmaxf(0, sqrtf(x_1) + sigma*sqrtf(psi_minus_k)*y/2); // X1
                        };
                    };
                    x_1 *= expo_2;
                } else {
                    u_1 = hiprand_uniform(&localState);
                    u_tilde_1 = x_1 * expo_2 + aux_u_tilde_1;
                    u_tilde_2 = u_tilde_1 * u_tilde_1 + sigma * sigma * psi_k  * (a * psi_k  / 2.0f + x_1 * expo_2);
                    u_tilde_3 = u_tilde_1 * u_tilde_2 + sigma_2 * psi_k * (x_1*x_1 * aux_u_tilde_3_2 + aux_u_tilde_3_1* (3*x_1 * expo_2 + a * psi_k));
                    s = (u_tilde_3 - u_tilde_1 * u_tilde_2) / (u_tilde_2 - u_tilde_1 * u_tilde_1);
                    p = (u_tilde_1 * u_tilde_3 - u_tilde_2*u_tilde_2) / (u_tilde_2 - u_tilde_1*u_tilde_1);
                    delta = sqrtf(s*s - 4.*p);
                    pi = (u_tilde_1 - (s-delta) / 2.) / delta;
                    if (u_1 < pi) {
                        x_1 = (s + delta) / 2;
                    } else {
                        x_1 = (s - delta) / 2;
                    };
                };
/////////////// Fin CIR_O3
                dx_1 += x_1;
                x_2 += (x_1 - 0.5*dx_1) * dt;
                x_4 += 0.5*x_3*dt;
                x_3 = x_3 * expf( aux_hw_1 + aux_hw_2 * dx_1 + aux_hw_3 * (x_1 - 0.5 * dx_1) );
                x_4 += 0.5*x_3*dt;
/////////////// HZ
                x_3 = x_3 * expf(sqrtf(x_1 * aux_hz_1)*n.x);
            }
        };
        if(type == 'e'){
            value_option = fmaxf(0, strike - x_3);
        }
        else {
            value_option = fmaxf(0, strike - x_4);
        };

        partial_sums[threadIdx.x] += value_option;
        partial_sums_squared[threadIdx.x] += value_option * value_option;
        // printf("%f\n", value_option);
    };

    // Synchronize the threads
    __syncthreads();

    // Sum per block
    if(threadIdx.x == 0){
        float sum = 0;
        float sum_squared = 0.0f;
        for (int i = 0; i < blockDim.x; ++i){
            sum += partial_sums[i];
            sum_squared += partial_sums_squared[i];
        };
        results_sum[blockIdx.x] += sum;
        results_sum_squared[blockIdx.x] += sum_squared;
    };

    // Saving the states in the global memory
    states[id] = localState;
};

void wrapper_kernel_o3(float* output, float x_0, float cir_0, float r, float a, float k, float sigma, float rho, float expiry, float strike, unsigned int num_steps, char type){
    clock_t t1;
    clock_t t2;
    long elapsed;

    t1 = clock();
    float *h_results_sum, *d_results_sum;
    h_results_sum = (float*)malloc(num_blocks * sizeof(float));
    hipMalloc(&d_results_sum, num_blocks * sizeof(float));
    for(int i = 0; i < num_blocks; ++i){
        h_results_sum[i] = 0;
    };
    float *h_results_sum_squared, *d_results_sum_squared;
    h_results_sum_squared = (float*)malloc(num_blocks * sizeof(float));
    hipMalloc(&d_results_sum_squared, num_blocks * sizeof(float));
    for(int i = 0; i < num_blocks; ++i){
        h_results_sum_squared[i] = 0;
    };
    hipMemcpy(d_results_sum, h_results_sum, num_blocks * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_results_sum_squared, h_results_sum_squared, num_blocks * sizeof(float), hipMemcpyHostToDevice);

    hiprandState *d_states;
    hipMalloc(&d_states, num_threads * num_blocks * sizeof(hiprandState));

    setup_states <<< num_blocks, num_threads >>> (d_states);
    hipDeviceSynchronize();
    put_heston_cir_o3 <<< num_blocks, num_threads >>> (d_states, d_results_sum, d_results_sum_squared, x_0, cir_0, r, a, k, sigma, rho, expiry, strike, num_steps, type);
    hipDeviceSynchronize();

    hipMemcpy(h_results_sum, d_results_sum, num_blocks * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_results_sum_squared, d_results_sum_squared, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Global sum
    float global_results_sum = 0;
    for(int i = 0; i < num_blocks; i++){
        global_results_sum += h_results_sum[i];
    };
    global_results_sum = global_results_sum * expf(-r * expiry);
    float global_results_sum_squared = 0;
    for(int i = 0; i < num_blocks; i++){
        global_results_sum_squared += h_results_sum_squared[i];
    };
    global_results_sum_squared = global_results_sum_squared * expf(- 2 * r * expiry);
    unsigned int num_simu = (num_blocks * num_threads * num_iterations);
    float empirical_expectency = global_results_sum / num_simu;
    float empirical_squared = global_results_sum_squared / num_simu;
    float empirical_variance = empirical_squared - empirical_expectency * empirical_expectency;
    float confidence_interval_low = empirical_expectency - 1.96 * sqrtf(empirical_variance / num_simu);
    float confidence_interval_high = empirical_expectency + 1.96 * sqrtf(empirical_variance / num_simu);

    t2 = clock();
    elapsed = timediff(t1, t2);

    output[0] = empirical_expectency;
    output[1] = confidence_interval_low;
    output[2] = elapsed;

    printf("We have computed a MC Call price of : %f\n", empirical_expectency);
    printf("Empirical variance : %f\n", empirical_variance);
    printf("Number of simulations: %d\n", num_simu);
    printf("Confidence interval : (%f , %f)\n", confidence_interval_low, confidence_interval_high);
    printf("Time elapsed: %ld ms\n", elapsed);

    return;
};

void cuda_plot_graph_performance(double expiry, double strike, double cir_0, double x_0, double a, double k , double sigma, double rho,double r, char type, int* num_steps_array, int num_points, unsigned int cap, double precision, double exact_value){

    float output_2[3];
    float output_3[3];

    FILE *f = fopen("plot.dat", "w");
    if (f == NULL) {
        printf("Error opening file!\n");
        exit(1);
    };
    unsigned int num_steps;
    for (unsigned int i = 0; i < num_points; ++i){
        num_steps = num_steps_array[i];
        wrapper_kernel_o2(output_2, x_0, cir_0, r, a, k, sigma, rho, expiry, strike, num_steps, type);
        wrapper_kernel_o3(output_3, x_0, cir_0, r, a, k, sigma, rho, expiry, strike, num_steps, type);
        fprintf(f, "%f\t%f\t%f\t%f\t%f\n", 1./num_steps, output_2[0], output_2[1], output_3[0], output_3[1]);
    };
    printf("We have computed the trajectories.\n");
    fclose(f);
    printf("We have written the data.");
    *f = fopen("gnu", "w");
    fprintf(f,"set nokey\n") ;
    fprintf(f, "set xlabel \"Inverse of number of steps\"\n");
    fprintf(f, "plot ");
    fprintf(f, "\"plot.dat\" using 1:2:3 with yerrorlines, \\\n");
    fprintf(f, "\"plot.dat\" using 1:4:5 with yerrorlines, \\\n");
    fprintf(f, "with lines lt 3");
    fclose(f);

    printf("We have written the gnuplot file.\n");

    return;
};

int main(void){
    float k = 0.5f;
    float a = 0.02f;
    float sigma = 0.4f;
    float x_0 = 100.0f;
    float cir_0 = 0.04;
    float rho = - 0.5f;
    float r = 0.02f;
    float strike = 100.0f;
    float expiry = 1.0f;
    unsigned int num_steps = 100;

    float output[3];
    wrapper_kernel_o2(output, x_0, cir_0, r, a, k, sigma, rho, expiry, strike, num_steps, 'e');
    wrapper_kernel_o3(output, x_0, cir_0, r, a, k, sigma, rho, expiry, strike, num_steps, 'e');

    printf("%f\t%f\t%f", output[0], output[1], output[2]);
    return 0;
}
